#include "hip/hip_runtime.h"
/**
 * @author smh
 * @date 2018.11.06
 *
 * @brief test Shfl instruction in CUDA.
 */

#include "testShflCUDA.h"
#include <iostream>

using namespace std;

__global__ void bcast(int arg)
{
    // 这里可以看出来，一个warp是对应于同一个block而言的，因为threadIdx对不同的block是独立的
    int laneId = threadIdx.x & 0x1f;   // 等价于对32取余，即只保留低5位，高位被强制置零
    int value;
    if(laneId == 0)                    // 每个warp的第一个thread
        value = arg;

    // 也就是说，__shfl指令返回的是源thread中var变量的值
    value = __shfl(value, 0);          // shfl return a 4-byte word referenced by var from the source lane ID as an unsigned integer.
    if(value != arg)
        printf("Thread %d failed.\n", threadIdx.x);
}

__global__ void testShfl_scan4()
{
	int laneId = threadIdx.x & 0x1F;     // caller's lane ID

	int value = 31 - laneId;

	for(int i = 1; i <= 4; i *= 2)
	{
		// 这里的n就是从源lane中value的数值
		// 这里使用__shfl_up实现了从左到右的scan4s算法
		// 使用_shfl_down实现从右到左的scane4算法
		int n = __shfl_up(value, i, 8);   // return the value of var held by the resulting lane ID！lower delta lanes will be unchanged.
		if(laneId >= i)
			value += n;
        else  // 当laneId超过warpSize的范围(小于0或大于warpSize参数)时，返回的值就是源lane的value值
            printf("thread %d the returned value is: %d \n", threadIdx.x, n);  // 返回的结果就是源laneId的value值
	}

	printf("Thread %d final value = %d \n", threadIdx.x, value);
}

// reduction across a warp
__global__ void testShfl_warpReduce()
{
	int laneId = threadIdx.x & 0x1F;

	int value = 31 - laneId;

	// use XOR mode to perform butterfly reduction
	for(int i = 16; i >= 1; i/=2)
        // 这里以butterfly addression pattern 的方式实现了32个线程(warp)内的Reduction
        // butterfly addression pattern的stride从i=16, 8, 4, 2, 1递减的，最后的结果就是全局Reduction，且warp里面各个线程的计算结果都是相同的。
		value += __shfl_xor(value, i, 32);  // calculate a source line ID by performing a bitwise XOR of the caller's lane ID with laneMask: the value of var held by the resulting (source) lane ID is returned.

	printf("Thread %d final value = %d \n", threadIdx.x, value);
}

void testShfl::testShflKernel()
{
    // 输入的数据是2个矩阵，每个矩阵的尺寸为16行32列
    float *data_d, *data_h = new float[32 * 32];  // 32 * 16 * 2
    float *data_out_h = new float [32 * 32];
    for (int i = 0; i < 32 * 32; ++i)
        data_h[i] = i;
    /**
    for (int i = 0; i < 16; ++i)    // 行
        for (int j = 0; j < 32; ++j)  // 列
        {
            int idx = i * 32 + j;
            data_h[idx] = 0;
        }
    for (int i = 0; i < 16; ++i)    // 行
        for (int j = 0; j < 32; ++j)  // 列
        {
            int idx = i * 32 + j;
            data_h[idx] = 1;
        }
    */

    hipError_t cudaStatus = hipSuccess;
    size_t d_pitch = 0;
    cudaStatus = hipMallocPitch(&data_d, &d_pitch, 32, 32);
    assert(cudaStatus == hipSuccess);
    cudaStatus = hipMemcpy2D(data_d, d_pitch, data_h, 32 * sizeof(float), 32 * sizeof(float), 32, hipMemcpyHostToDevice);
    assert(cudaStatus == hipSuccess);

    dim3 blockPerGrid(2, 1, 1);
    dim3 threadPerBlock(32 * 16, 1, 1);
    //testShule<<<blockPerGrid, threadPerBlock>>>(data_d, data_h);
    //bcast<<<1, 32>>>(1234);
    //testShfl_scan4<<<1, 32>>>();
    testShfl_warpReduce<<<1, 32>>>();
    hipDeviceSynchronize();
    cout << hipGetErrorString(hipPeekAtLastError()) << endl;

    cudaStatus = hipMemcpy2D(data_out_h, 32 * sizeof(float), data_d, d_pitch, 32 * sizeof(float), 32, hipMemcpyDeviceToHost);
    assert(cudaStatus == hipSuccess);

    cudaStatus = hipFree(data_d);
    assert(cudaStatus == hipSuccess);

    delete [] data_h;
    delete [] data_out_h;

}

