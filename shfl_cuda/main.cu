#include <iostream>
#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void testShfl(int arg)
{
	int laneIdx = threadIdx.x & 0x1F;

	int value = 0;
	if(laneIdx == 0)
		value = arg;

	value = __shfl(value, 0);  // return the 4-byte word referenced by var from the source lane ID as an unsigned integer.
	if(value != arg)
		printf("Thread %d failed.\n", threadIdx.x);
}

__global__ void testShfl_scan4()
{
	int laneId = threadIdx.x & 0x1F;     // caller's lane ID

	int value = 31 - laneId;

	for(int i = 1; i <= 4; i *= 2)
	{
		// 这里的n就是从源lane中value的数值
		// 这里使用__shfl_up实现了从左到右的scan4s算法
		// 使用_shfl_down实现从右到左的scane4算法
		int n = __shfl_up(value, i, 8);   // return the value of var held by the resulting lane ID！lower delta lanes will be unchanged.
		if(laneId >= i)
			value += n;
	}

	printf("Thread %d final value = %d \n", threadIdx.x, value);
}

// reduction across a warp
__global__ void testShfl_warpReduce()
{
	int laneId = threadIdx.x & 0x1F;

	int value = 31 - laneId;

	// use XOR mode to perform butterfly reduction
	for(int i = 16; i >= 1; i/=2)
		value += __shfl_xor(value, i, 32);  // calculate a source line ID by performing a bitwise XOR of the caller's lane ID with laneMask: the value of var held by the resulting (source) lane ID is returned.

	printf("Thread %d final value = %d \n", threadIdx.x, value);
}


using namespace std;

int main()
{
	//testShfl<<<1, 32>>>(1234);
	testShfl_warpReduce<<<1, 32>>>();

	hipDeviceSynchronize();
	cout << hipGetErrorString(hipPeekAtLastError()) << endl;

	return 0;
}
